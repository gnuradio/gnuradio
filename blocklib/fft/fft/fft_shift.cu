
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

template <typename T>
__global__ void kernel_fft_shift(const T* in, T* out, int n);

template <>
__global__ void kernel_fft_shift(const hipFloatComplex* in, hipFloatComplex* out, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        float a = 1 - 2 * (i & 1);
        out[i].x = in[i].x * a;
        out[i].y = in[i].y * a;
    }
}

template <>
__global__ void kernel_fft_shift(const hipDoubleComplex* in, hipDoubleComplex* out, int n)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < n) {
        double a = 1 - 2 * (i & 1);
        out[i].x = in[i].x * a;
        out[i].y = in[i].y * a;
    }
}


void exec_fft_shift(const hipFloatComplex* in,
                    hipFloatComplex* out,
                    int n,
                    int grid_size,
                    int block_size,
                    hipStream_t stream)
{
    kernel_fft_shift<<<grid_size, block_size, 0, stream>>>(in, out, n);
}
