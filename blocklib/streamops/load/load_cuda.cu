/* -*- c++ -*- */
/*
 * Copyright 2021 Josh Morman
 *
 * This file is part of GNU Radio
 *
 * SPDX-License-Identifier: GPL-3.0-or-later
 *
 */

#include <hip/hip_complex.h>
#include <hip/hip_runtime.h>


// The block cuda file is just a wrapper for the kernels that will be launched in the work
// function
namespace gr {
namespace streamops {
namespace load_cu {
    __global__ void load_kernel(const uint8_t* in, uint8_t* out, int N, size_t load = 1)
    {
        int i = blockIdx.x * blockDim.x + threadIdx.x;
        if (i < N) {
            for (int x = 0; x < load; x++) {
                out[i] = in[i];
            }
        }
    }
    
    void exec_kernel(const uint8_t* in,
                     uint8_t* out,
                     int grid_size,
                     int block_size,
                     int N,
                     size_t load,
                     hipStream_t stream)
    {
        load_kernel<<<grid_size, block_size, 0, stream>>>(in, out, N, load);
    }
    

void get_block_and_grid(int* minGrid, int* minBlock)
{
    hipOccupancyMaxPotentialBlockSize(minGrid, minBlock, load_kernel, 0, 0);
}

} // namespace load_cu
} // namespace streamops
} // namespace gr