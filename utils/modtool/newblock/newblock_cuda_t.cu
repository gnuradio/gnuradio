#include "hip/hip_runtime.h"
/* -*- c++ -*- */
/*
 * Copyright <COPYRIGHT_YEAR> <COPYRIGHT_AUTHOR>
 *
 * This file is part of GNU Radio
 *
 * SPDX-License-Identifier: GPL-3.0-or-later
 *
 */

#include "newblock_cuda.cuh"
#include <hip/hip_complex.h>
#include <complex>

namespace gr {
namespace newmod {
namespace newblock_cu {

template <typename T>
__global__ void newblock_kernel(const T* in, T* out, T k, size_t n)
{
    // Do block specific code here
}

template <>
__global__ void newblock_kernel(const hipFloatComplex* in,
                                      hipFloatComplex* out,
                                      hipFloatComplex k,
                                      size_t n)
{
    // Do block specific code here
}

template <typename T>
void exec_kernel(
    const T* in, T* out, T k, int grid_size, int block_size, hipStream_t stream)
{
    newblock_kernel<T>
        <<<grid_size, block_size, 0, stream>>>(in, out, k, block_size * grid_size);
}

template <typename T>
void get_block_and_grid(int* minGrid, int* minBlock)
{
    // https://developer.nvidia.com/blog/cuda-pro-tip-occupancy-api-simplifies-launch-configuration/
    hipOccupancyMaxPotentialBlockSize(minGrid, minBlock, newblock_kernel<T>, 0, 0);
}

#define IMPLEMENT_KERNEL(T)                                                            \
    template void exec_kernel<T>(                                                      \
        const T* in, T* out, T k, int grid_size, int block_size, hipStream_t stream); \
    template void get_block_and_grid<T>(int* minGrid, int* minBlock);


IMPLEMENT_KERNEL(int16_t)
IMPLEMENT_KERNEL(int32_t)
IMPLEMENT_KERNEL(float)
IMPLEMENT_KERNEL(hipFloatComplex)

} // namespace newblock_cu
} // namespace newmod
} // namespace gr