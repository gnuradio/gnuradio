#include "hip/hip_runtime.h"
/* -*- c++ -*- */
/*
 * Copyright <COPYRIGHT_YEAR> <COPYRIGHT_AUTHOR>
 *
 * This file is part of GNU Radio
 *
 * SPDX-License-Identifier: GPL-3.0-or-later
 *
 */

 #include "newblock_cuda.cuh"

// The block cuda file is just a wrapper for the kernels that will be launched in the work
// function
namespace gr {
namespace newmod {
namespace newblock_cu {
__global__ void newblock_kernel(const uint8_t* in, uint8_t* out, int batch_size)
{
    // block specific code goes here
}

void exec_newblock(
    const uint8_t* in, uint8_t* out, int grid_size, int block_size, hipStream_t stream)
{
    int batch_size = block_size * grid_size;
    newblock_kernel<<<grid_size, block_size, 0, stream>>>(in, out, batch_size);
}

void get_block_and_grid(int* minGrid, int* minBlock)
{
    // https://developer.nvidia.com/blog/cuda-pro-tip-occupancy-api-simplifies-launch-configuration/
    hipOccupancyMaxPotentialBlockSize(minGrid, minBlock, newblock_kernel, 0, 0);
}
} // namespace newblock_cu
} // namespace newmod
} // namespace gr